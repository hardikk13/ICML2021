#include "denseLayer.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_batched.h>
#include <cutlass/epilogue/thread/linear_combination_relu.h>
#include <cutlass/gemm/device/default_gemm_configuration.h>


// Aliases
using ColumnMajor = cutlass::layout::ColumnMajor;
using ColumnMajorTransposed = cutlass::layout::RowMajor;

using ArchTag = cutlass::arch::Sm60;
using OpClass = cutlass::arch::OpClassSimt;

using relu_op = cutlass::epilogue::thread::LinearCombinationRelu<float,1>;
using linear_op = cutlass::epilogue::thread::LinearCombination<float,1>;

// Use Gemm defaults (for now...)
using DefaultConfig = cutlass::gemm::device::DefaultGemmConfiguration<OpClass, //op class
                                                                ArchTag, // arch tag
                                                                float,  // element a
                                                                float,  // element b
                                                                float,  // element c
                                                                float>; // element accum

using ThreadblockShape = cutlass::gemm::GemmShape<8,32,8>;
using WarpShape = cutlass::gemm::GemmShape<8,16,8>;


using GemmRelu = cutlass::gemm::device::Gemm<
                                                float,                      // Data-type of A matrix
                                                ColumnMajorTransposed,      // Layout of A matrix
                                                float,                      // Data-type of B matrix
                                                ColumnMajor,                // Layout of B matrix
                                                float,                      // Data-type of C matrix
                                                ColumnMajor,                // Layout of C matrix                                                             
                                                float,                      // Element Accumulator Type
                                                OpClass,
                                                ArchTag,
                                                ThreadblockShape, 
                                                WarpShape, 
                                                DefaultConfig::InstructionShape,
                                                relu_op>;

using GemmLinear = cutlass::gemm::device::Gemm<
                                                float, ColumnMajorTransposed,  
                                                float, ColumnMajor,     
                                                float, ColumnMajor,                                                                 
                                                float,        
                                                OpClass,
                                                ArchTag,
                                                ThreadblockShape, 
                                                WarpShape, 
                                                DefaultConfig::InstructionShape,
                                                linear_op>;

using BatchedGemmRelu = cutlass::gemm::device::GemmBatched<
                                                float, ColumnMajorTransposed,
                                                float, ColumnMajor,
                                                float, ColumnMajor,
                                                float, 
                                                OpClass,
                                                ArchTag,
                                                ThreadblockShape, 
                                                WarpShape, 
                                                DefaultConfig::InstructionShape,
                                                relu_op>;


using BatchedGemmLinear = cutlass::gemm::device::GemmBatched<
                                                float, ColumnMajorTransposed,
                                                float, ColumnMajor,
                                                float, ColumnMajor,
                                                float,
                                                OpClass,
                                                ArchTag,
                                                ThreadblockShape, 
                                                WarpShape, 
                                                DefaultConfig::InstructionShape,
                                                linear_op>;


hipError_t denseLayerForward(
    float* W, float* A, float* Z, float* b, 
    int M, int N, int K,
    int activation) {

    cutlass::cutStatus status;

    if (activation == ReLU) {
        GemmRelu gemm;
        // Construct the CUTLASS GEMM arguments object.
        GemmRelu::Arguments args({M , N, K},        // Gemm Problem dimensions
            {W, K},                                 // ref for source matrix A
            {A, K},                                 // ref for source matrix B
            {b, M},                                 // ref for source matrix C
            {Z, M},                                 // ref for destination matrix D
            {1.0f, 1.0f});                          // Scalars used in the Epilogue
         status = gemm(args);
    } else {
        GemmLinear gemm;
        // Construct the CUTLASS GEMM arguments object.
        GemmLinear::Arguments args({M , N, K},
            {W, K},    
            {A, K},           
            {b, M},            
            {Z, M},             
            {1.0f, 1.0f});     
        status = gemm(args);
    }
          
    // Return a hipError_t if the CUTLASS GEMM operator returned an error code.
    if (status != cutlass::cutStatus::kSuccess) {
        printf("ERROR: %s\n", cutlassGetStatusString(status));
        printf("Single - M: %d N: %d K: %d\n", M,N,K);
        return hipErrorUnknown;
    }

    // Return success, if no errors were encountered.
    return hipSuccess;
}

hipError_t batchedDenseLayerForward(
    const float* W, const float* A, float* Z, const float* b, 
    int M, int N, int K, int numBatches,
    int activation)
 {

    cutlass::cutStatus status;

    if (activation == ReLU) {
        BatchedGemmRelu batchedGemm;

        status = batchedGemm({
            {M, N, K},
            {W, K}, 
            0,
            {A, K}, 
            K,        // batch in N
            {b, M}, 
            0,
            {Z, M}, 
            M,        // stride
            {1.0f, 1.0f},
            numBatches
        });
    } else {
        BatchedGemmLinear batchedGemm;

        status = batchedGemm({
            {M, N, K},
            {W, K}, 
            0,
            {A, K}, 
            K,        // stride
            {b, M}, 
            0,
            {Z, M}, 
            M,        // stride
            {1.0f, 1.0f},
            numBatches
        });
    }
    
    if (status != cutlass::cutStatus::kSuccess) {
        printf("ERROR: %s\n", cutlassGetStatusString(status));
        printf("Batched - M: %d N: %d K: %d numBatches: %d\n", M,N,K, numBatches);
        return hipErrorUnknown;
    }

    return hipSuccess;

 }


// Dense Layer Class imp
DenseLayer::DenseLayer(
    std::string name, 
    std::vector<std::vector<float>> weights, 
    std::vector<float> biases,
    int activation, bool hostOnly
) 
{
    this->W = Matrix(Shape(weights.size(), weights[0].size()), hostOnly);
    this->numWeightParams = weights.size() * weights[0].size();
    this->b = Matrix(Shape(biases.size(), 1), hostOnly); // (1,B)
    this->numBiasParams = biases.size();
    this->name = name;
    this->type = eDense;
    this->hostOnly = hostOnly;

    b.allocateMemory();
    W.allocateMemory();
    
    initializeBias(biases);
    initializeWeights(weights);

    this->activation = activation;
}

DenseLayer::~DenseLayer()
{ }

void DenseLayer::initializeBias(std::vector<float> biases) {
    for (int x = 0; x < biases.size(); x ++) {
        b[x] = biases[x];
    }

    if (!hostOnly) {
        b.copyHostToDevice();
    }
}

void DenseLayer::initializeWeights(std::vector<std::vector<float>> weights) {
    for (int x = 0; x < weights.size(); x++) {
        for (int y = 0; y < weights[0].size(); y ++) {
            W[y*W.shape.x + x] = weights[x][y]; //ROW MAJOR!
        }
    }

    if (!hostOnly) {
        W.copyHostToDevice();
    }
}

void Print(Matrix& matrix) {
    matrix.copyDeviceToHost();
    for (int i = 0; i < matrix.size(); ++i) {
       printf("\t %d, %f \n", i , matrix[i]);
    }
    std::cout << "\n\n";
}


Matrix& DenseLayer::forward(Matrix& A, int maxBatchSize) {
    assert(W.shape.x == A.shape.x);

    this->A = A;

    Shape Z_shape;
    if (maxBatchSize == -1) {
        Z_shape = Shape(W.shape.y, A.shape.y);
    } else {
        Z_shape = Shape(W.shape.y, maxBatchSize);
    }
    
    Z.maybeAllocateMemory(Z_shape);

    hipError_t ok = computeAndStoreLayerOutput(A);
    checkCudaErrors(ok);

    Z.shape.y = A.shape.y;
    // Print(Z);
    return Z;   
}

hipError_t DenseLayer::computeAndStoreLayerOutput(Matrix& A) {
    hipError_t ok;

    if (A.shape.y == 1) {
        // single item
        ok = denseLayerForward(
            W.deviceData.get(), A.deviceData.get(), Z.deviceData.get(), b.deviceData.get(), 
            W.shape.y, // transpose!
            b.shape.y, 
            W.shape.x, 
            this->activation
        );
    } else {
        // batched !
        ok = batchedDenseLayerForward(
            (const float*)W.deviceData.get(), (const float*)A.deviceData.get(), Z.deviceData.get(), (const float*)b.deviceData.get(), 
            W.shape.y,  //M
            b.shape.y,  //N
            W.shape.x,  //K
            A.shape.y,  //Num Batches
            this->activation
        );
    }

    checkCudaErrors(ok);

    return hipSuccess;
}

int DenseLayer::getXDim() const {
	return W.shape.x;
}

int DenseLayer::getYDim() const {
	return W.shape.y;
}

Matrix DenseLayer::getWeightsMatrix() const {
	return W;
}

Matrix DenseLayer::getBiasVector() const {
	return b;
}

